#include "hip/hip_runtime.h"
// ------------------------------------------------------------------
// Faster R-CNN
// Copyright (c) 2015 Microsoft
// Licensed under The MIT License [see fast-rcnn/LICENSE for details]
// Written by Shaoqing Ren
// ------------------------------------------------------------------

#include "quad_gpu_nms.hpp"
#include <vector>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define CUDA_CHECK(condition) \
  /* Code block avoids redefinition of hipError_t error */ \
  do { \
    hipError_t error = condition; \
    if (error != hipSuccess) { \
      std::cout << hipGetErrorString(error) << std::endl; \
    } \
  } while (0)

#define DIVUP(m,n) ((m) / (n) + ((m) % (n) > 0))
int const threadsPerBlock = sizeof(unsigned long long) * 14;

__device__ inline float quad_devIoU(float const * const ba, float const * const ax, float const * const ay, float const * const bb, float const * const bx, float const * const by) {
  int xmin,ymin,xmax,ymax;
  if (ba[0]<=bb[0]) xmin = (int)ba[0]; else xmin = (int)bb[0];
  if (ba[2]>=bb[2]) xmax = (int)ba[2]; else xmax = (int)bb[2];
  if (ba[1]<=bb[1]) ymin = (int)ba[1]; else ymin = (int)bb[1];
  if (ba[3]>=bb[3]) ymax = (int)ba[3]; else ymax = (int)bb[3];

  float b_width = bb[2]-bb[0], b_height = bb[3]-bb[1]; 
  float a_width = ba[2]-ba[0], a_height = ba[3]-ba[1];
  if(((xmax-xmin)>=(a_width+b_width)) || ((ymax-ymin)>=(a_height+b_height)))
    return 0;
  
  bool oddNodes = false; //
  int countOverlap=0; int countGT=0; int countPB=0;
  for(float i=xmin; i<=xmax; i=i+(xmax-xmin)*0.01) 
  {
    for(float j=ymin; j<=ymax; j=j+(ymax-ymin)*0.01)
    { 
      int k,l = 14;
      oddNodes=false;
      for (k =0; k<14;k++){
        if((ay[k] < j && ay[l] >= j || ay[l] < j && ay[k] >= j) && (ax[k] <= i || ax[l] <= i)){
          oddNodes^=(ax[k]+(j-ay[k])/(ay[l]-ay[k])*(ax[l]-ax[k])<i);
        }
      }
      countPB+=int(oddNodes);
      if (oddNodes==true)
      {
        oddNodes=false;
        int k,l = 14;
        for (k =0; k<14;k++){
        if((by[k] < j && by[l] >= j || by[l] < j && by[k] >= j) && (bx[k] <= i || bx[l] <= i)){
          oddNodes^=(bx[k]+(j-by[k])/(by[l]-by[k])*(bx[l]-bx[k])<i);
          }
        }
        countGT+=int(oddNodes);
        countOverlap+=int(oddNodes);
      }
      else
      {
        oddNodes=false;
        int k,l = 14;
        for (k =0; k<14;k++){
        if((by[k] < j && by[l] >= j || by[l] < j && by[k] >= j) && (bx[k] <= i || bx[l] <= i)){
          oddNodes^=(bx[k]+(j-by[k])/(by[l]-by[k])*(bx[l]-bx[k])<i);
          }
        }
        countGT+=int(oddNodes);
      }
    } 
  }
  return (countOverlap)*1.0/(countPB+countGT-countOverlap);  
}

__global__ void nms_kernel(const int n_boxes, const float nms_overlap_thresh,
                           const float *bdev_boxes, const float *xdev_boxes, const float *ydev_boxes, unsigned long long *dev_mask) {
  const int row_start = blockIdx.y;
  const int col_start = blockIdx.x;


  const int row_size =
        min(n_boxes - row_start * threadsPerBlock, threadsPerBlock);
  const int col_size =
        min(n_boxes - col_start * threadsPerBlock, threadsPerBlock);

  __shared__ float xblock_boxes[threadsPerBlock * 14]; 
  __shared__ float yblock_boxes[threadsPerBlock * 14]; 
  __shared__ float bblock_boxes[threadsPerBlock * 14]; 

  if (threadIdx.x < col_size) {
    for (int j = 0; j<14; j++){
      bblock_boxes[threadIdx.x * 14 + j] = 
        bdev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 14 + j];   
      xblock_boxes[threadIdx.x * 14 + j] = 
        xdev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 14 + j];   
      yblock_boxes[threadIdx.x * 14 + j] = 
        ydev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 14 + j];   
    }
  }
  __syncthreads();

  if (threadIdx.x < row_size) {
    const int cur_box_idx = threadsPerBlock * row_start + threadIdx.x;
    const float *bcur_box = bdev_boxes + cur_box_idx * 14; 
    const float *xcur_box = xdev_boxes + cur_box_idx * 14; 
    const float *ycur_box = ydev_boxes + cur_box_idx * 14; 
    int i = 0;
    unsigned long long t = 0;
    int start = 0;
    if (row_start == col_start) {
      start = threadIdx.x + 1;
    }
    for (i = start; i < col_size; i++) {
      // float sc = quad_devIoU(bcur_box, xcur_box, ycur_box, bblock_boxes + i*14, xblock_boxes + i * 14, yblock_boxes + i * 14);
      // printf("%f\n", 1.0);
      if (quad_devIoU(bcur_box, xcur_box, ycur_box, bblock_boxes + i*14, xblock_boxes + i * 14, yblock_boxes + i * 14) > nms_overlap_thresh) { 
        t |= 1ULL << i;
      }
    }
    const int col_blocks = DIVUP(n_boxes, threadsPerBlock);
    dev_mask[cur_box_idx * col_blocks + col_start] = t;
  }
}

void _set_device(int device_id) {
  int current_device;
  CUDA_CHECK(hipGetDevice(&current_device));
  if (current_device == device_id) {
    return;
  }
  // The call to hipSetDevice must come before any calls to Get, which
  // may perform initialization using the GPU.
  CUDA_CHECK(hipSetDevice(device_id));
}

void _nms_quad(int* keep_out, int* num_out, const float* boxes_bound, const float* boxes_hostx, const float* boxes_hosty, int boxes_num,  int boxes_dim, float nms_overlap_thresh, int device_id) {
  _set_device(device_id);

  float* boxes_devx = NULL;
  float* boxes_devy = NULL;
  float* boxes_devb = NULL;
  unsigned long long* mask_dev = NULL;

  const int col_blocks = DIVUP(boxes_num, threadsPerBlock);
  // std::cout<<"col_blocks: "<<col_blocks<<std::endl;  // 1
  // std::cout<<"sizeof(unsigned long long): "<<sizeof(unsigned long long)<<std::endl;  // 8
  CUDA_CHECK(hipMalloc(&boxes_devx,
                        boxes_num * boxes_dim * sizeof(float)));
  CUDA_CHECK(hipMalloc(&boxes_devy,
                        boxes_num * boxes_dim * sizeof(float)));
  CUDA_CHECK(hipMalloc(&boxes_devb,
                        boxes_num * boxes_dim * sizeof(float)));

  CUDA_CHECK(hipMemcpy(boxes_devx,
                        boxes_hostx,
                        boxes_num * boxes_dim * sizeof(float),
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(boxes_devy,
                        boxes_hosty,
                        boxes_num * boxes_dim * sizeof(float),
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(boxes_devb,
                        boxes_bound,
                        boxes_num * boxes_dim * sizeof(float),
                        hipMemcpyHostToDevice));

  CUDA_CHECK(hipMalloc(&mask_dev,
                        boxes_num * col_blocks * sizeof(unsigned long long)));

  dim3 blocks(DIVUP(boxes_num, threadsPerBlock),
              DIVUP(boxes_num, threadsPerBlock));
  dim3 threads(threadsPerBlock);

  int bbb; std::cout<<"bbb"<<std::endl; std::cin>>bbb;
  nms_kernel<<<blocks, threads>>>(boxes_num,
                                  nms_overlap_thresh,
                                  boxes_devb,
                                  boxes_devx,
                                  boxes_devy,
                                  mask_dev);
  
  int bbb1; std::cout<<"bbb1"<<std::endl; std::cin>>bbb1;
  std::vector<unsigned long long> mask_host(boxes_num * col_blocks);
  // unsigned long long* mask_host = NULL;
  // CUDA_CHECK(hipHostMalloc(&mask_host,
                            // boxes_num * col_blocks * sizeof(unsigned long long)));
  int bbb2; std::cout<<"bbb2 "<<mask_dev[0]<<std::endl; std::cin>>bbb2;
  CUDA_CHECK(hipMemcpy(&mask_host[0],
                        mask_dev,
                        sizeof(unsigned long long) * boxes_num * col_blocks,
                        hipMemcpyDeviceToHost)); // an illegal instruction was encountered
  std::vector<unsigned long long> remv(col_blocks);
  memset(&remv[0], 0, sizeof(unsigned long long) * col_blocks);
  int num_to_keep = 0;
  for (int i = 0; i < boxes_num; i++) {
    int nblock = i / threadsPerBlock;
    int inblock = i % threadsPerBlock;
    // int aaa=1; std::cout<<"threadsPerBlock "<< threadsPerBlock<< std::endl
    //                     <<"boxes_num "<< boxes_num<< std::endl
    //                     <<"inblock "<< inblock<< std::endl
    //                     <<"(1ULL << inblock) "<< (1ULL << inblock)<< std::endl
    //                     <<"remv[nblock] "<< remv[nblock]<< std::endl
    //                     <<"nblock "<< nblock<< std::endl
    //                     <<"col_blocks "<< col_blocks<< std::endl;
    //                     std::cin>>aaa;
    if (!(remv[nblock] & (1ULL << inblock))) {
      keep_out[num_to_keep++] = i;
      unsigned long long *p = &mask_host[0] + i * col_blocks;
      for (int j = nblock; j < col_blocks; j++) {
        remv[j] |= p[j];
      }
    }
  }
  int ddd; std::cout<<"ddd"<<std::endl; std::cin>>ddd;
  *num_out = num_to_keep;
  int eee; std::cout<<"eee"<<std::endl; std::cin>>eee;
  CUDA_CHECK(hipFree(boxes_devb)); // an illegal instruction was encountered
  CUDA_CHECK(hipFree(boxes_devx)); // an illegal instruction was encountered
  CUDA_CHECK(hipFree(boxes_devy)); // an illegal instruction was encountered
  CUDA_CHECK(hipFree(mask_dev));
  int fff; std::cout<<"fff"<<std::endl; std::cin>>fff;
}
